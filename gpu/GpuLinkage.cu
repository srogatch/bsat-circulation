#include "GpuLinkage.cuh"

void HostLinkage::Init(const Formula& formula, const CudaAttributes& ca) {
  pFormula_ = &formula;
  pCa_ = &ca;
  nVars_ = pFormula_->nVars_;
  nClauses_ = pFormula_->nClauses_;
  headsClause2Var_ = CudaArray<GpuPerSignHead>(nClauses_*2 + 2, CudaArrayType::Device);
  headsVar2Clause_ = CudaArray<GpuPerSignHead>(nVars_*2 + 2, CudaArrayType::Device);
  std::unique_ptr<GpuPerSignHead[]> h_headsClause2Var = std::make_unique<GpuPerSignHead[]>(nClauses_ * 2 + 2);
  std::unique_ptr<GpuPerSignHead[]> h_headsVar2Clause = std::make_unique<GpuPerSignHead[]>(nVars_ * 2 + 2);

  VciGpu total;

  // Populate heads for clauses
  total = 0;
  for(VciGpu i=-pFormula_->nClauses_; i<=pFormula_->nClauses_; i++) {
    if(i == 0) {
      h_headsClause2Var[i+pFormula_->nClauses_][GpuLinkage::SignToHead(-1)] = total;  
      h_headsClause2Var[i+pFormula_->nClauses_][GpuLinkage::SignToHead(+1)] = total;
      continue;
    }
    h_headsClause2Var[i+pFormula_->nClauses_][GpuLinkage::SignToHead(-1)] = total;
    total += pFormula_->clause2var_.ArcCount(i, -1);
    h_headsClause2Var[i+pFormula_->nClauses_][GpuLinkage::SignToHead(+1)] = total;
    total += pFormula_->clause2var_.ArcCount(i, +1);
  }
  h_headsClause2Var[2*pFormula_->nClauses_+1][GpuLinkage::SignToHead(-1)] = total;  
  h_headsClause2Var[2*pFormula_->nClauses_+1][GpuLinkage::SignToHead(+1)] = total;
  gpuErrchk(hipMemcpyAsync(
    headsClause2Var_.Get(), h_headsClause2Var.get(), sizeof(GpuPerSignHead) * size_t(2*nClauses_+2),
    hipMemcpyHostToDevice, pCa_->cs_
  ));
  targetsClause2Var_ = CudaArray<VciGpu>(total, CudaArrayType::Device);
  std::unique_ptr<VciGpu[]> h_targetsClause2Var = std::make_unique<VciGpu[]>(total);

  // Populate targets for clauses
  total = 0;
  for(VciGpu i=-pFormula_->nClauses_; i<=pFormula_->nClauses_; i++) {
    if(i == 0) {
      continue;
    }
    for(int8_t sign=-1; sign<=1; sign+=2) {
      const VciGpu enJ = pFormula_->clause2var_.ArcCount(i, sign);
      for(VciGpu j=0; j<enJ; j++) {
        h_targetsClause2Var[total] = pFormula_->clause2var_.GetTarget(i, sign, j);
        total++;
      }
    }
  }
  gpuErrchk(hipMemcpyAsync(
    targetsClause2Var_.Get(), h_targetsClause2Var.get(), sizeof(VciGpu) * total,
    hipMemcpyHostToDevice, pCa_->cs_
  ));

  // Populate heads for vars
  total = 0;
  for(VciGpu i=-pFormula_->nVars_; i<=pFormula_->nVars_; i++) {
    if(i == 0) {
      h_headsVar2Clause[i+pFormula_->nVars_][GpuLinkage::SignToHead(-1)] = total;  
      h_headsVar2Clause[i+pFormula_->nVars_][GpuLinkage::SignToHead(+1)] = total;
      continue;
    }
    h_headsVar2Clause[i+pFormula_->nVars_][GpuLinkage::SignToHead(-1)] = total;
    total += pFormula_->var2clause_.ArcCount(i, -1);
    h_headsVar2Clause[i+pFormula_->nVars_][GpuLinkage::SignToHead(+1)] = total;
    total += pFormula_->var2clause_.ArcCount(i, +1);
  }
  h_headsVar2Clause[2*pFormula_->nVars_+1][GpuLinkage::SignToHead(-1)] = total;  
  h_headsVar2Clause[2*pFormula_->nVars_+1][GpuLinkage::SignToHead(+1)] = total;
  gpuErrchk(hipMemcpyAsync(
    headsVar2Clause_.Get(), h_headsVar2Clause.get(), sizeof(GpuPerSignHead) * size_t(nVars_ * 2 + 2),
    hipMemcpyHostToDevice, pCa_->cs_
  ));
  targetsVar2Clause_ = CudaArray<VciGpu>(total, CudaArrayType::Device);
  std::unique_ptr<VciGpu[]> h_targetsVar2Clause = std::make_unique<VciGpu[]>(total);

  // Populate targets for vars
  total = 0;
  for(VciGpu i=-pFormula_->nVars_; i<=pFormula_->nVars_; i++) {
    if(i == 0) {
      continue;
    }
    for(int8_t sign=-1; sign<=1; sign+=2) {
      const VciGpu enJ = pFormula_->var2clause_.ArcCount(i, sign);
      for(VciGpu j=0; j<enJ; j++) {
        h_targetsVar2Clause[total] = pFormula_->var2clause_.GetTarget(i, sign, j);
        total++;
      }
    }
  }
  gpuErrchk(hipMemcpyAsync(
    targetsVar2Clause_.Get(), h_targetsVar2Clause.get(), sizeof(VciGpu) * total,
    hipMemcpyHostToDevice, pCa_->cs_
  ));

  // We must synchronize here because the host memory on the stack of this function will be released upon function's exit,
  // while the stream is still copying this host memory into GPU.
  gpuErrchk((hipStreamSynchronize(pCa_->cs_)));
}

bool HostLinkage::Marshal(GpuLinkage& gl) {
  gl.headsClause2Var_ = headsClause2Var_.Get();
  gl.headsVar2Clause_ = headsVar2Clause_.Get();
  gl.nClauses_ = nClauses_;
  gl.nVars_ = nVars_;
  gl.targetsClause2Var_ = targetsClause2Var_.Get();
  gl.targetsVar2Clause_ = targetsVar2Clause_.Get();
  return true;
}
