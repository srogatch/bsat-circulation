#include "hip/hip_runtime.h"
//#undef NDEBUG

#include "Common.h"

#include <cassert>

constexpr const uint32_t kThreadsPerBlock = 128;
constexpr const uint8_t kL2SolRoundRobin = 13; // log2( # solutions in the round-robin )
constexpr const uint32_t kMaxVarFrontSize = 4096;

#include "CpuInit.h"

#include "GpuLinkage.cuh"
__constant__ GpuLinkage gLinkage;

#include "GpuRainbow.cuh"
__constant__ GpuRainbow gSeenAsgs;

#include "GpuConstants.cuh"
// This must be included after gpHashSeries is defined
#include "GpuBitVector.cuh"
#include "GpuTraversal.cuh"
#include "GpuTrackingVector.cuh"

struct SystemShared {
  GpuTraversal trav_;
  __uint128_t *solRRasgs_;
  VciGpu *solRRnsUnsat_;
  VciGpu nGlobalUnsat_;
  VciGpu nUnsatExecs_;
  VciGpu firstSolRR_;
  VciGpu limitSolRR_;
  int syncRR_;

  __device__ void Record(const GpuBitVector& asg, const VciGpu nUnsat) {
    bool full;
    VciGpu target = -1;
    for(;;) {
      // Lock
      while(atomicCAS_system(&syncRR_, 0, 1) != 0) {
        __nanosleep(256);
      }
      const VciGpu newLimit = (limitSolRR_ + 1) & ((VciGpu(1)<<kL2SolRoundRobin)-1);
      full = (newLimit == firstSolRR_);
      if(!full) {
        target = limitSolRR_;
        limitSolRR_ = newLimit;
        // Lock
        atomicExch_system(solRRnsUnsat_ + target, -1);
      }
      // Unlock
      atomicExch_system(&syncRR_, 0);
      if(!full) {
        break;
      }
      __nanosleep(1024);
    }
    assert(!full);
    assert( 0 <= target && target < (VciGpu(1)<<kL2SolRoundRobin) );
    __uint128_t *pWrite = solRRasgs_ + uint64_t(target) * asg.VectCount();
    for(VciGpu i=0; i<asg.VectCount(); i++) {
      pWrite[i] = reinterpret_cast<__uint128_t*>(asg.bits_)[i];
    }
    // Unlock
    atomicExch_system(solRRnsUnsat_+target, nUnsat);
  }

  __host__ bool Consume(BitVector& asg, VciGpu &nUnsat) volatile {
    VciGpu iPop = -1;
    static_assert(std::atomic<int>::is_always_lock_free, "Must be same size as int");
    volatile std::atomic<int> *pSync = reinterpret_cast<volatile std::atomic<int>*>(&syncRR_);
    
    // Lock
    int state = 0;
    while(!pSync->compare_exchange_strong(state, 1)) {
      assert(state == 1);
      state = 0;
      __builtin_ia32_pause();
    }

    const VciGpu iFirst = firstSolRR_;
    if(iFirst != limitSolRR_) {
      iPop = iFirst;
      firstSolRR_ = (iFirst + 1) & ((VciGpu(1)<<kL2SolRoundRobin)-1);
    }
    
    // Unlock
    pSync->store(0);

    if(iPop == -1) {
      return false;
    }

    volatile std::atomic<VciGpu>* pCounterLock = reinterpret_cast<volatile std::atomic<VciGpu>*>(solRRnsUnsat_ + iPop);
    while( (nUnsat = pCounterLock->load()) == -1 ) {
      __builtin_ia32_pause();
    }
    memcpy(asg.bits_.get(), solRRasgs_ + uint64_t(iPop) * DivUp(asg.nBits_, 128), asg.nQwords_ * sizeof(uint64_t));
    return true;
  }
};

struct GpuExec {
  Xoshiro256ss rng_; // seed it on the host
  GpuBitVector nextAsg_; // nVars+1 bits
  GpuUnordSet unsatClauses_;
  // GpuTrackingVector<VciGpu> front_;
  VciGpu *varFrontItems_;
  VciGpu varFrontSize_;
};

struct PerGpuInfo {
  CudaArray<GpuExec> execs_;
  CudaArray<__uint128_t> bvBufs_;
  CudaArray<VciGpu> allVarFrontItems_;
  GpuLinkage gl_;
  GpuRainbow gr_;
  uint32_t nStepBlocks_;  
};

__global__ void ReplicateAssignment(GpuExec* execs, const uint32_t nExecs) {
  const uint32_t iThread = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t nThreads = blockDim.x * gridDim.x;
  for(uint32_t i=iThread + 1; i<nExecs; i+=nThreads) {
    GpuExec& curExec = execs[i];
    VectCopy(curExec.nextAsg_.bits_, execs[0].nextAsg_.bits_, curExec.nextAsg_.VectCount() * sizeof(__uint128_t));
    VciGpu iFlip = curExec.rng_.Next() % (curExec.nextAsg_.nBits_ - 1) + 1;
    curExec.nextAsg_.Flip(iFlip);
  }
}

__global__ void StepKernel(const VciGpu nStartUnsat, SystemShared* sysShar, GpuExec *execs)
{
  constexpr const uint32_t cCombsPerStep = 1u<<11;
  const uint32_t iThread = threadIdx.x + blockIdx.x * kThreadsPerBlock;
  assert(blockDim.x == kThreadsPerBlock);
  // const uint32_t nThreads = gridDim.x * kThreadsPerBlock;
  GpuExec& curExec = execs[iThread];

  if(curExec.unsatClauses_.buffer_ == nullptr) {
    assert(curExec.unsatClauses_.bitsPerPack_ == 0);
    assert(curExec.unsatClauses_.nBuckets_ == 0);
    assert(curExec.unsatClauses_.hash_ == 0);
    assert(curExec.unsatClauses_.count_ == 0);

    const VciGpu nClauses=gLinkage.GetClauseCount();
    curExec.unsatClauses_ = GpuUnordSet(nStartUnsat, nClauses);
    for(VciGpu i=1; i<=nClauses; i++) {
      if(!IsSatisfied(i, curExec.nextAsg_)) {
        curExec.unsatClauses_.Add(i);
      }
    }
  }

  assert(curExec.unsatClauses_.count_ >= sysShar->nGlobalUnsat_);

  while(curExec.unsatClauses_.count_ >= nStartUnsat && sysShar->nGlobalUnsat_ >= nStartUnsat) {
    // Save memory for varFront
    curExec.unsatClauses_.Shrink( curExec.rng_.Next() );
    // Get the variables that affect the unsatisfied clauses
    const GpuUnordSet& combClauses = curExec.unsatClauses_; // front_ ?
    curExec.varFrontSize_ = 0;
    combClauses.Visit<false>(curExec.rng_.Next(), [&](const VciGpu aClause) -> bool {
      for(int8_t sign=-1; sign<=1; sign+=2) {
        const VciGpu varListLen = gLinkage.ClauseArcCount(aClause, sign);
        for(VciGpu j=0; j<varListLen; j++) {
          const VciGpu iVar = gLinkage.ClauseGetTarget(aClause, sign, j);
          const VciGpu aVar = abs(iVar);
          // Let the duplicate variables appear multiple times in the array, and thus
          // be considered for combinations multiple times proportionally to their
          // entry numbers.
          assert(1 <= aVar && aVar <= gLinkage.GetVarCount());
          if(curExec.varFrontSize_ < kMaxVarFrontSize) {
            curExec.varFrontItems_[curExec.varFrontSize_] = aVar;
            curExec.varFrontSize_++;
          } else {
            return false;
            // Reservoir sampling
            // const uint64_t r = curExec.rng_.Next() % totVarFront;
            // if(r < kMaxVarFrontSize) {
            //   curExec.varFrontItems_[r] = aVar;
            // }
          }
        }
      }
      return true;
    });

    // Shuffle the front
    for(VciGpu i=0; i+1<curExec.varFrontSize_; i++) {
      const VciGpu pos = i + curExec.rng_.Next() % (curExec.varFrontSize_ - i);
      Swap(curExec.varFrontItems_[i], curExec.varFrontItems_[pos]);
    }

    //// Combine
    GpuTrackingVector<VciGpu> stepRevs;
    VciGpu bestUnsat = gLinkage.GetClauseCount() + 1;
    GpuTrackingVector<VciGpu> bestRevVars;
    // Make sure the overhead of preparing the combinations doesn't outnumber the effort spent in combinations
    uint32_t endComb = cCombsPerStep; //max(cCombsPerStep, totVarFront);
    if(curExec.varFrontSize_ <= 31) [[unlikely]] {
      endComb = min(endComb, (1u<<curExec.varFrontSize_)-1);
    }
    // Initial assignment
    uint32_t curComb = 1;
    {
      const VciGpu aVar = curExec.varFrontItems_[0];
      assert(1 <= aVar && aVar <= gLinkage.GetVarCount());
      stepRevs.Add<false>(aVar);
      curExec.nextAsg_.Flip(aVar);
      UpdateUnsatCs(aVar, curExec.nextAsg_, curExec.unsatClauses_);
    }
    // The first index participating in combinations - upon success, can be shifted
    VciGpu combFirst = 0;
    while(curComb <= endComb) {
      if(!sysShar->trav_.IsSeenAsg(curExec.nextAsg_)) {
        if(curExec.unsatClauses_.count_ < bestUnsat) {
          bestUnsat = curExec.unsatClauses_.count_;
          bestRevVars = stepRevs;
          // TODO: remove (DEBUG)
          // curExec.unsatClauses_.Visit([&](const VciGpu aClause) {
          //   assert( !IsSatisfied(aClause, curExec.nextAsg_) );
          // });
          // for(VciGpu i=1; i<=gLinkage.GetClauseCount(); i++) {
          //   if(IsSatisfied(i, curExec.nextAsg_)) {
          //     assert(!curExec.unsatClauses_.Contains(i));
          //   } else {
          //     assert(curExec.unsatClauses_.Contains(i));
          //   }
          // }
          if(bestUnsat < nStartUnsat) [[unlikely]] {
            const VciGpu oldMin = atomicMin_system(&sysShar->nGlobalUnsat_, bestUnsat);
            if(oldMin > bestUnsat) [[likely]] {
              sysShar->Record(curExec.nextAsg_, bestUnsat);
              combFirst = combFirst + 32 - __clz(curComb);
              curComb = 0;
              const VciGpu remVF = curExec.varFrontSize_ - combFirst;
              if(remVF <= 31) [[unlikely]] {
                endComb = min(cCombsPerStep, (1u<<remVF)-1);
              } else {
                endComb = cCombsPerStep;
              }
            }
          }
        }
        if(curExec.unsatClauses_.count_ <= sysShar->nGlobalUnsat_) [[unlikely]] {
          sysShar->trav_.RecordAsg(curExec.nextAsg_, curExec.unsatClauses_.count_);
        }
      }
      for(uint8_t i=0; ; i++) {
        curComb ^= 1u << i;
        assert(i + combFirst < curExec.varFrontSize_);
        const VCIndex aVar = curExec.varFrontItems_[i+combFirst];
        assert(1 <= aVar && aVar <= gLinkage.GetVarCount());
        [[maybe_unused]] const bool bExisted = stepRevs.Flip(aVar);
        // Variables may repeat inside varFront
        //assert( bExisted == !(curComb & (1u << i)) );
        curExec.nextAsg_.Flip(aVar);
        UpdateUnsatCs(aVar, curExec.nextAsg_, curExec.unsatClauses_);
        if( (curComb & (1u << i)) != 0u ) {
          break;
        }
      }
    }

    // Check the combinations results
    if(bestUnsat > gLinkage.GetClauseCount()) [[unlikely]] {
      if(sysShar->trav_.StepBack(curExec.nextAsg_, curExec.unsatClauses_, gLinkage.GetClauseCount())) [[likely]] {
        continue;
      }
      // Increment the unsatisfied executors counter
      atomicAdd_system(&sysShar->nUnsatExecs_, 1);
      // The current executor considers it unsatisfiable, but let's wait for the rest of executors
      break;
    }

    // Revert to the best assignment
    stepRevs.Sort();
    bestRevVars.Sort();
    VciGpu iSR = 0, iBR = 0;
    while(iSR < stepRevs.count_ || iBR < bestRevVars.count_) {
      assert(iSR == 0 || iSR >= stepRevs.count_ || stepRevs.items_[iSR-1] < stepRevs.items_[iSR]);
      assert(iBR == 0 || iBR >= bestRevVars.count_ || bestRevVars.items_[iBR-1] < bestRevVars.items_[iBR]);
      VciGpu aVar;
      if( iBR >= bestRevVars.count_ || (iSR < stepRevs.count_ && stepRevs.items_[iSR] < bestRevVars.items_[iBR]) ) {
        aVar = stepRevs.items_[iSR];
        iSR++;
      } else if( iSR >= stepRevs.count_ || (iBR < bestRevVars.count_ && bestRevVars.items_[iBR] < stepRevs.items_[iSR]) ) {
        aVar = bestRevVars.items_[iBR];
        iBR++;
      } else {
        assert(iSR < stepRevs.count_);
        assert(iBR < bestRevVars.count_);
        assert(stepRevs.items_[iSR] == bestRevVars.items_[iBR]);
        iSR++;
        iBR++;
        continue;
      }
      // if(!(1 <= aVar && aVar <= gLinkage.GetVarCount())) {
      //   printf(" %d ", aVar);
      // }
      curExec.nextAsg_.Flip(aVar);
      UpdateUnsatCs(aVar, curExec.nextAsg_, curExec.unsatClauses_);
    }
    // // TODO: remove (DEBUG)
    // for(VciGpu i=1; i<=gLinkage.GetClauseCount(); i++) {
    //   if(IsSatisfied(i, curExec.nextAsg_)) {
    //     assert(!curExec.unsatClauses_.Contains(i));
    //   } else {
    //     assert(curExec.unsatClauses_.Contains(i));
    //   }
    // }
    // // TODO: remove (DEBUG)
    // if(curExec.unsatClauses_.count_ != bestUnsat) {
    //   while(atomicCAS_system(&sysShar->syncRR_, 0, 1) != 0) {
    //     __nanosleep(256);
    //   }
    //   printf("stepRevs: ");
    //   for(VciGpu i=0; i<stepRevs.count_; i++) {
    //     printf(" %d ", stepRevs.items_[i]);
    //   }
    //   printf("\nbestRevVars: ");
    //   for(VciGpu i=0; i<bestRevVars.count_; i++) {
    //     printf(" %d ", bestRevVars.items_[i]);
    //   }
    //   printf("\n");
    //   atomicExch_system(&sysShar->syncRR_, 0);
    //   assert(false);
    // }
    assert(curExec.unsatClauses_.count_ == bestUnsat);
    assert(curExec.unsatClauses_.count_ >= sysShar->nGlobalUnsat_);

    if(sysShar->nGlobalUnsat_ < nStartUnsat) [[unlikely]] {
      break; // some other executor found an improvement
    }

    // TODO: Sequential Gradient Descent
  }
}

int main(int argc, char* argv[]) {
  auto tmStart = std::chrono::steady_clock::now();
  const auto tmVeryStart = tmStart;

  if(argc < 3) {
    std::cerr << "Usage: " << argv[0] << " <input.dimacs> <output.dimacs> [<RamGBs>]" << std::endl;
    return 1;
  }

  uint64_t maxRamBytes = 0;
  if(argc >= 4) {
    maxRamBytes = std::stoull(argv[3]) * 1024 * uint64_t(1024) * 1024;
  }
  if(maxRamBytes == 0) {
    maxRamBytes = GetTotalSystemMemory() * 0.95;
  }
  
  // TODO: does it override the environment variable?
  omp_set_num_threads(nSysCpus);
  const uint64_t nOmpThreads = omp_get_max_threads();

  Formula formula;
  std::atomic<bool> provenUnsat = false;
  std::atomic<bool> maybeSat = formula.Load(argv[1]);
  if(!maybeSat) {
    provenUnsat = true;
    { // TODO: remove code duplication
      std::ofstream ofs(argv[2]);
      ofs << "s UNSATISFIABLE" << std::endl;
      // TODO: output the proof: proof.out, https://satcompetition.github.io/2024/output.html
    }
    return 0;
  }
  int64_t prevNUnsat = formula.nClauses_;

  std::cout << "Precomputing..." << std::endl;
  int nGpus = 0;
  gpuErrchk(hipGetDeviceCount(&nGpus));
  std::vector<CudaAttributes> cas(nGpus);
  std::vector<CudaArray<__uint128_t>> gpuHSes;
  std::vector<HostRainbow> seenAsgs(nGpus); // must be one per device

  for(int i=0; i<nGpus; i++) {
    cas[i].Init(i);
  }
  GpuCalcHashSeries(std::max(formula.nVars_, formula.nClauses_), cas, gpuHSes);

  // TODO: call CPU Init here
  const VciGpu bestInitNUnsat = CpuInit(formula);

  std::cout << "Preparing GPU data structures" << std::endl;
  std::vector<HostLinkage> linkages;
  HostLinkage::Init(formula, cas, linkages);
  std::vector<CudaArray<GpuExec>> execs(nGpus);
  std::vector<PerGpuInfo> pgis(nGpus);
  const VciGpu nVectsPerVarsBV = DivUp(formula.nVars_ + 1, 128);
  // BPCT - Bytes Per CUDA Thread
  const uint64_t hostHeapBpct
    = sizeof(GpuExec)
    + nVectsPerVarsBV * sizeof(__uint128_t) // GpuExec::nextAsg_
    + kMaxVarFrontSize * sizeof(VciGpu) // varFrontItems_
    + 256 * 3 // Alignment
    + 256; // Thread stack
  const uint64_t deviceHeapBpct
    // GpuExec::unsatClauses_
    // varFront
    // stepRevs
    // bestRevVars
    = ( (bestInitNUnsat / GpuUnordSet::cStartOccupancy + 16) * ceilf(log2f(formula.nClauses_+1)) / 8
    + kMaxVarFrontSize * 2 * sizeof(VciGpu) + 16 * 6 );
  const uint64_t overheadBpct = deviceHeapBpct/6;

    // Pinned should be better than managed here, because managed memory transfers at page granularity,
  // while Pinned - at PCIe bus granularity, which is much smaller.
  CudaArray<SystemShared> sysShar(1, CudaArrayType::Managed);
  HostPartSolDfs dfsAsg;
  dfsAsg.Init( maxRamBytes / 2, formula.ans_.nBits_ );
  CudaArray<VciGpu> solRRnsUnsat( 1u<<kL2SolRoundRobin, CudaArrayType::Managed );
  
  #pragma omp parallel for num_threads(nGpus)
  for(int i=0; i<nGpus; i++) {
    gpuErrchk(hipSetDevice(i));
    linkages[i].Marshal(pgis[i].gl_);
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(gLinkage), &pgis[i].gl_, sizeof(pgis[i].gl_), 0, hipMemcpyHostToDevice, cas[i].cs_));
    int nBlocksPerSM = 0;
    gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&nBlocksPerSM, &StepKernel, kThreadsPerBlock, 0));
    // This is the upper bound for now without the correction for the actually available VRAM
    pgis[i].nStepBlocks_ = nBlocksPerSM * cas[i].cdp_.multiProcessorCount;
    gpuErrchk(hipMemGetInfo(&cas[i].freeBytes_, &cas[i].totalBytes_));
    uint64_t maxRainbowBytes = 1ULL << lround(ceil(log2(cas[i].freeBytes_/3)));
    for(;;) {
      uint64_t bytesBothHeaps = pgis[i].nStepBlocks_ * uint64_t(kThreadsPerBlock) * (hostHeapBpct + deviceHeapBpct + overheadBpct);
      uint64_t rainbowBytes = 1ULL << int(std::log2(maxRainbowBytes));
      uint64_t totVramReq = bytesBothHeaps + rainbowBytes;
      if(totVramReq <= cas[i].freeBytes_) {
        break;
      }
      const double reduction = std::sqrt( double(cas[i].freeBytes_) / totVramReq );
      maxRainbowBytes *= reduction;
      pgis[i].nStepBlocks_ *= reduction;
    }
    seenAsgs[i].Init(maxRainbowBytes, cas[i]);
    gpuErrchk(hipMemGetInfo(&cas[i].freeBytes_, &cas[i].totalBytes_));
    pgis[i].nStepBlocks_ = std::min<uint64_t>(
      nBlocksPerSM * cas[i].cdp_.multiProcessorCount,
      cas[i].freeBytes_
        / (uint64_t(kThreadsPerBlock) * (hostHeapBpct + deviceHeapBpct + overheadBpct))
    );
    Logger() << "Rainbow Table: " << double(uint64_t(seenAsgs[i].nbfDwords_) * sizeof(uint32_t)) / (1ULL<<30)
      << " GB, Host heap: "
      << double(pgis[i].nStepBlocks_) *  kThreadsPerBlock * hostHeapBpct / (1ULL<<30)
      << " GB, Device heap: "
      << double(pgis[i].nStepBlocks_) *  kThreadsPerBlock * deviceHeapBpct / (1ULL<<30)
      << " GB. nStepBlocks: " << pgis[i].nStepBlocks_;

    // Enable dynamic memory allocation in the CUDA kernel
    gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize,
      pgis[i].nStepBlocks_ * uint64_t(kThreadsPerBlock) * deviceHeapBpct));
    pgis[i].bvBufs_ = CudaArray<__uint128_t>(
      pgis[i].nStepBlocks_ * uint64_t(kThreadsPerBlock) * nVectsPerVarsBV, CudaArrayType::Device
    );
    pgis[i].allVarFrontItems_ = CudaArray<VciGpu>(
      pgis[i].nStepBlocks_ * uint64_t(kThreadsPerBlock) * kMaxVarFrontSize, CudaArrayType::Device
    );
    pgis[i].execs_ = CudaArray<GpuExec>(
      pgis[i].nStepBlocks_ * uint64_t(kThreadsPerBlock), CudaArrayType::Device
    );
  }
  std::atomic<int64_t> totExecs = 0;
  std::vector<std::unique_ptr<GpuExec[]>> vCpuExecs(nGpus);
  #pragma omp parallel for num_threads(nGpus)
  for(int i=0; i<nGpus; i++) {
    gpuErrchk(hipSetDevice(i));
    vCpuExecs[i] = std::make_unique<GpuExec[]>(pgis[i].execs_.Count());
    totExecs.fetch_add(pgis[i].execs_.Count());
    std::random_device rd;
    for(uint32_t j=0; j<pgis[i].execs_.Count(); j++) {
      for(int k=0; k<int(sizeof(vCpuExecs[i][j].rng_.s_)); k+=sizeof(uint32_t)) {
        reinterpret_cast<uint32_t*>(&vCpuExecs[i][j].rng_.s_)[k/sizeof(uint32_t)] = rd();
      }
      vCpuExecs[i][j].nextAsg_.hash_ = formula.ans_.hash_;
      vCpuExecs[i][j].nextAsg_.nBits_ = formula.nVars_ + 1;
      vCpuExecs[i][j].nextAsg_.bits_ = reinterpret_cast<uint32_t*>(
        pgis[i].bvBufs_.Get() + nVectsPerVarsBV * uint64_t(j));
      vCpuExecs[i][j].varFrontItems_ = pgis[i].allVarFrontItems_.Get() + uint64_t(j) * kMaxVarFrontSize;
      assert(vCpuExecs[i][j].unsatClauses_.buffer_ == nullptr);
    }
    // TODO: tail bits (beyond the last QWord, but withing the last 128-bit vector) may be corrupt
    gpuErrchk(hipMemcpyAsync(vCpuExecs[i][0].nextAsg_.bits_, formula.ans_.bits_.get(),
      formula.ans_.nQwords_ * sizeof(uint64_t), hipMemcpyHostToDevice, cas[i].cs_
    ));
    gpuErrchk(hipMemcpyAsync(
      pgis[i].execs_.Get(), vCpuExecs[i].get(), pgis[i].execs_.Count() * sizeof(GpuExec),
      hipMemcpyHostToDevice, cas[i].cs_
    ));
    seenAsgs[i].Marshal(pgis[i].gr_);
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(
      gSeenAsgs), &pgis[i].gr_, sizeof(pgis[i].gr_), 0, hipMemcpyHostToDevice, cas[i].cs_));
  }
  #pragma omp parallel for num_threads(nGpus)
  for(int i=0; i<nGpus; i++) {
    gpuErrchk(hipSetDevice(i));
    int nBlocksPerSM = 0;
    gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&nBlocksPerSM, &ReplicateAssignment, kThreadsPerBlock, 0));
    // This is the upper bound for now without the correction for the actually available VRAM
    int totBlocks = nBlocksPerSM * cas[i].cdp_.multiProcessorCount;
    ReplicateAssignment<<<totBlocks, kThreadsPerBlock, 0, cas[i].cs_>>>(
      pgis[i].execs_.Get(), pgis[i].execs_.Count());
  }
  #pragma omp parallel for num_threads(nGpus)
  for(int i=0; i<nGpus; i++) {
    gpuErrchk(hipSetDevice(i));
    gpuErrchk(hipStreamSynchronize(cas[i].cs_));
    vCpuExecs[i].reset(); // release the memory
  }
  vCpuExecs.clear();

  sysShar.Get()->trav_.dfsAsg_ = dfsAsg.Marshal();
  sysShar.Get()->trav_.syncDfs_ = 0;
  sysShar.Get()->nGlobalUnsat_ = bestInitNUnsat;
  sysShar.Get()->nUnsatExecs_ = 0;
  CudaArray<__uint128_t> solRRasgs( (1u<<kL2SolRoundRobin) * uint64_t(nVectsPerVarsBV), CudaArrayType::Pinned );
  sysShar.Get()->firstSolRR_ = sysShar.Get()->limitSolRR_ = 0;
  sysShar.Get()->solRRasgs_ = solRRasgs.Get();
  sysShar.Get()->solRRnsUnsat_ = solRRnsUnsat.Get();
  sysShar.Get()->syncRR_ = 0;

  std::cout << "Running on GPU(s)" << std::endl;
  std::atomic<VciGpu> bestNUnsat = bestInitNUnsat;
  std::thread solUpdater([&] {
    BitVector asg(formula.nVars_ + 1);
    VciGpu nUnsat;
    while(bestNUnsat > 0) {
      if(!sysShar.Get()->Consume(asg, nUnsat)) {
        std::this_thread::sleep_for(std::chrono::microseconds(10));
        continue;
      }
      if(nUnsat < bestNUnsat) {
        bestNUnsat = nUnsat;
        formula.ans_ = asg;
      }
    }
  });

  VciGpu nStartUnsat = bestNUnsat;
  auto reportStats = [&] {
    auto tmEnd = std::chrono::steady_clock::now();
    nStartUnsat = bestNUnsat;
    double nSec = std::chrono::duration_cast<std::chrono::nanoseconds>(tmEnd - tmStart).count() / 1e9;
    double clausesPerSec = (prevNUnsat - nStartUnsat) / nSec;
    std::cout << "\n\tUnsatisfied clauses: " << nStartUnsat << " - elapsed " << nSec << " seconds, ";
    if(clausesPerSec >= 1 || clausesPerSec == 0) {
      std::cout << clausesPerSec << " clauses per second.";
    } else {
      std::cout << 1.0 / clausesPerSec << " seconds per clause.";
    }
    std::cout << " Time since very start: "
      << std::chrono::duration_cast<std::chrono::nanoseconds>(tmEnd - tmVeryStart).count() / (60 * 1e9)
      << " minutes." << std::endl;
    tmStart = tmEnd;
    prevNUnsat = nStartUnsat;
  };

  while(bestNUnsat > 0) {
    reportStats();
    #pragma omp parallel for num_threads(nGpus)
    for(int i=0; i<nGpus; i++) {
      gpuErrchk(hipSetDevice(i));
      assert(pgis[i].nStepBlocks_ * kThreadsPerBlock == pgis[i].execs_.Count());
      StepKernel<<<pgis[i].nStepBlocks_, kThreadsPerBlock, 0, cas[i].cs_>>>(
        nStartUnsat, sysShar.Get(), pgis[i].execs_.Get() );
      gpuErrchk(hipGetLastError());
    }
    #pragma omp parallel for num_threads(nGpus)
    for(int i=0; i<nGpus; i++) {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipStreamSynchronize(cas[i].cs_));
    }
    if(sysShar.Get()->nUnsatExecs_ == totExecs) {
      maybeSat = false;
      break;
    }
    sysShar.Get()->nUnsatExecs_ = 0;
  }
  reportStats();
  solUpdater.join();

  if(nStartUnsat == 0) {
    std::cout << "SATISFIED" << std::endl;
  } else if(maybeSat) {
    std::cout << "UNKNOWN" << std::endl;
  } else {
    std::cout << "UNSATISFIABLE" << std::endl;
  }
  return 0;
}
