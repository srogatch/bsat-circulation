#include "hip/hip_runtime.h"
#include "Common.h"

#include "GpuLinkage.cuh"

constexpr const uint32_t kThreadsPerBlock = 128;

__constant__ __uint128_t *gpHashSeries;
std::unique_ptr<uint128[]> BitVector::hashSeries_ = nullptr;

void GpuCalcHashSeries(const VciGpu maxItem, const std::vector<CudaAttributes>& cas) {
  BitVector::hashSeries_ = std::make_unique<uint128[]>(maxItem + 1);
  BitVector::hashSeries_[0] = 1;
  for(VciGpu i=1; i<=maxItem; i++) {
    BitVector::hashSeries_[i] = BitVector::hashSeries_[i-1] * kHashBase;
  }
  for(int i=0; i<int(cas.size()); i++) {
    gpuErrchk(hipSetDevice(i));
    gpuErrchk(hipMemcpyToSymbolAsync(HIP_SYMBOL(gpHashSeries), BitVector::hashSeries_.get(), sizeof(__uint128_t)*(maxItem+1), 0,
      hipMemcpyHostToDevice, cas[i].cs_));
  }
  for(int i=0; i<int(cas.size()); i++) {
    gpuErrchk(hipSetDevice(i));
    gpuErrchk(hipStreamSynchronize(cas[i].cs_));
  }
}

// This must be included after gpHashSeries is defined
#include "GpuBitVector.cuh"

struct Hasher {
  __uint128_t hash_;

  Hasher(const VciGpu item) {
    hash_ = item * kHashBase + 37;
  }
};

template<typename TItem> struct GpuTrackingVector {
  __uint128_t hash_ = 0;
  TItem* items_ = nullptr;
  VciGpu count_ = 0, capacity_ = 0;

  __host__ __device__ GpuTrackingVector() = default;

  // Returns whether the vector was resized
  __host__ __device__ bool Reserve(const VciGpu newCap) {
    if(newCap <= capacity_) {
      return false;
    }
    VciGpu maxCap = max(capacity_, newCap);
    capacity_ = maxCap + (maxCap>>1) + 16;
    items_ = realloc(items_, capacity_ * sizeof(TItem));
    return true;
  }

  // Returns whether the item existed in the collection
  __host__ __device__ bool Flip(const TItem item) {
    hash_ ^= Hasher(item).hash_;
    for(VciGpu i=0; i<count_; i++) {
      if(items_[i] == item) {
        items_[i] = items_[count_-1];
        count_--;
        return true;
      }
    }
    Reserve(count_+1);
    items_[count_] = item;
    count_++;
    return false;
  }

  // Add without checking whether such item already exists
  __host__ __device__ void Add(const TItem item) {
    hash_ ^= Hasher(item).hash_;
    Reserve(count_+1);
    items_[count_] = item;
    count_++;
  }

  __host__ __device__ ~GpuTrackingVector() {
    free(items_);
    #ifndef NDEBUG
    items_ = nullptr;
    #endif // NDEBUG
  }
};

struct GpuExec {
  GpuBitVector<true, false> next_;
  GpuTrackingVector<VciGpu> unsatClauses_;
  GpuTrackingVector<VciGpu> front_;
};

__global__ void StepKernel(const GpuLinkage linkage, GpuExec *execs, const uint64_t maxCombs) {
  const uint32_t iThread = threadIdx.x + blockIdx.x *  kThreadsPerBlock;
  const uint32_t nThreads = gridDim.x * kThreadsPerBlock;
  GpuExec& curExec = execs[iThread];
  
}

int main(int argc, char* argv[]) {
  auto tmStart = std::chrono::steady_clock::now();
  const auto tmVeryStart = tmStart;

  if(argc < 3) {
    std::cerr << "Usage: " << argv[0] << " <input.dimacs> <output.dimacs>" << std::endl;
    return 1;
  }
  
  // TODO: does it override the environment variable?
  omp_set_num_threads(nSysCpus);
  const uint64_t nOmpThreads = omp_get_max_threads();

  Formula formula;
  std::atomic<bool> provenUnsat = false;
  std::atomic<bool> maybeSat = formula.Load(argv[1]);
  if(!maybeSat) {
    provenUnsat = true;
    { // TODO: remove code duplication
      std::ofstream ofs(argv[2]);
      ofs << "s UNSATISFIABLE" << std::endl;
      // TODO: output the proof: proof.out, https://satcompetition.github.io/2024/output.html
    }
    return 0;
  }
  int64_t prevNUnsat = formula.nClauses_;

  std::cout << "Precomputing..." << std::endl;
  int nGpus = 0;
  gpuErrchk(hipGetDeviceCount(&nGpus));
  std::vector<CudaAttributes> cas(nGpus);
  for(int i=0; i<nGpus; i++) {
    cas[i].Init(i);
  }
  GpuCalcHashSeries(std::max(formula.nVars_, formula.nClauses_), cas);


  return 0;
}
