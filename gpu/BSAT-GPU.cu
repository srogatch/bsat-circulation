#include "hip/hip_runtime.h"
#include "Common.h"

#include "GpuLinkage.cuh"
#include "GpuConstants.cuh"
// This must be included after gpHashSeries is defined
#include "GpuBitVector.cuh"
#include "GpuTraversal.cuh"

constexpr const uint32_t kThreadsPerBlock = 128;

struct Hasher {
  __uint128_t hash_;

  __host__ __device__ Hasher(const VciGpu item) {
    hash_ = item * kHashBase + 37;
  }
};

template<typename TItem> struct GpuTrackingVector {
  __uint128_t hash_ = 0;
  TItem* items_ = nullptr;
  VciGpu count_ = 0, capacity_ = 0;

  GpuTrackingVector() = default;

  __host__ __device__ GpuTrackingVector(const GpuTrackingVector& src) {
    hash_ = src.hash_;
    count_ = src.count_;
    free(items_);
    capacity_ = src.count_;
    items_ = malloc(capacity_ * sizeof(TItem));
    // TODO: vectorize
    for(VciGpu i=0; i<count_; i++) {
      items_[i] = src.items_[i];
    }
  }

  __host__ __device__ GpuTrackingVector& operator=(const GpuTrackingVector& src) {
    if(this != &src) {
      hash_ = src.hash_;
      count_ = src.count_;
      if(capacity_ < src.count_) {
        free(items_);
        capacity_ = src.count_;
        items_ = reinterpret_cast<TItem*>(malloc(capacity_ * sizeof(TItem)));
      }
      // TODO: vectorize
      for(VciGpu i=0; i<count_; i++) {
        items_[i] = src.items_[i];
      }
    }
    return *this;
  }

  // Returns whether the vector was resized
  __host__ __device__ bool Reserve(const VciGpu newCap) {
    if(newCap <= capacity_) {
      return false;
    }
    VciGpu maxCap = max(capacity_, newCap);
    capacity_ = maxCap + (maxCap>>1) + 16;
    TItem* newItems = reinterpret_cast<TItem*>(malloc(capacity_ * sizeof(TItem)));
    // TODO: vectorize
    for(VciGpu i=0; i<count_; i++) {
      newItems[i] = items_[i];
    }
    free(items_);
    items_ = newItems;
    return true;
  }

  // Returns whether the item existed in the collection
  __host__ __device__ bool Flip(const TItem item) {
    hash_ ^= Hasher(item).hash_;
    for(VciGpu i=count_-1; i>=0; i--) {
      if(items_[i] == item) {
        items_[i] = items_[count_-1];
        count_--;
        return true;
      }
    }
    Reserve(count_+1);
    items_[count_] = item;
    count_++;
    return false;
  }

  // Returns whether a new item was added, or a duplicate existed
  template<bool checkDup> __host__ __device__ bool Add(const TItem item) {
    if constexpr(checkDup) {
      for(VciGpu i=count_-1; i>=0; i--) {
        if(items_[i] == item) {
          return false;
        }
      }
    }
    hash_ ^= Hasher(item).hash_;
    Reserve(count_+1);
    items_[count_] = item;
    count_++;
    return true;
  }

  // Returns true if the item had existed in the collection
  __host__ __device__ bool Remove(const TItem& item) {
    for(VciGpu i=count_-1; i>=0; i--) {
      if(items_[i] == item) {
        hash_ ^= Hasher(item).hash_;
        items_[i] = items_[count_-1];
        return true;
      }
    }
    return false;
  }

  __host__ __device__ ~GpuTrackingVector() {
    free(items_);
    #ifndef NDEBUG
    items_ = nullptr;
    #endif // NDEBUG
  }

  __host__ __device__ void Clear() {
    hash_ = 0;
    count_ = 0;
  }
};

struct GpuExec {
  Xoshiro256ss rng_; // seed it on the host
  GpuBitVector next_;
  GpuTrackingVector<VciGpu> unsatClauses_;
  // GpuTrackingVector<VciGpu> front_;
};

__device__ void UpdateUnsatCs(const GpuLinkage& linkage, const VciGpu aVar, const GpuBitVector& next,
  GpuTrackingVector<VciGpu>& unsatClauses)
{
  const int8_t signSat = next[aVar];
  const VciGpu nSatArcs = linkage.VarArcCount(aVar, signSat);
  for(VciGpu i=0; i<nSatArcs; i++) {
    const VciGpu iClause = linkage.VarGetTarget(aVar, signSat, i);
    const VciGpu aClause = abs(iClause);
    unsatClauses.Remove(aClause);
  }
  const VciGpu nUnsatArcs = linkage.VarArcCount(aVar, -signSat);
  for(VciGpu i=0; i<nUnsatArcs; i++) {
    const VciGpu iClause = linkage.VarGetTarget(aVar, -signSat, i);
    const VciGpu aClause = abs(iClause);
    unsatClauses.Add<true>(aClause);
  }
}

__global__ void StepKernel(const VciGpu nStartUnsat, VciGpu* pnGlobalUnsat, const GpuLinkage linkage, GpuExec *execs,
  GpuTraversal* trav, const GpuBitVector maxPartial, VciGpu* pnUnsatExecs)
{
  constexpr const uint32_t cCombsPerStep = 1u<<11;
  const uint32_t iThread = threadIdx.x + blockIdx.x *  kThreadsPerBlock;
  const uint32_t nThreads = gridDim.x * kThreadsPerBlock;
  GpuExec& curExec = execs[iThread];

  while(curExec.unsatClauses_.count_ >= nStartUnsat && *pnGlobalUnsat >= nStartUnsat) {
    // Get the variables that affect the unsatisfied clauses
    GpuTrackingVector<VciGpu> varFront;
    uint32_t totListLen = 0;
    const GpuTrackingVector<VciGpu>& combClauses = curExec.unsatClauses_; // front_ ?
    for(VciGpu i=0; i<combClauses.count_; i++) {
      for(int8_t sign=-1; sign<=1; sign+=2) {
        const VciGpu aClause = combClauses.items_[i];
        const VciGpu varListLen = linkage.ClauseArcCount(aClause, sign);
        totListLen += varListLen;
        for(VciGpu j=0; j<varListLen; j++) {
          const VciGpu iVar = linkage.ClauseGetTarget(aClause, sign, j);
          const VciGpu aVar = abs(iVar);
          if( curExec.next_[aVar] != Signum(iVar) ) {
            varFront.Add<false>(aVar);
            // TODO: this is incorrect - the same variable may appear with the opposite sign in another clause
            curExec.next_.Flip(aVar);
          }
        }
      }
    }
    // Flip back the marked vars
    for(VciGpu i=0; i<varFront.count_; i++) {
      curExec.next_.Flip(varFront.items_[i]);
    }
    // Shuffle the front
    for(VciGpu i=0; i<varFront.count_; i++) {
      const VciGpu pos = i + curExec.rng_.Next() % (varFront.count_ - i);
      const VciGpu t = varFront.items_[i];
      varFront.items_[i] = varFront.items_[pos];
      varFront.items_[pos] = t;
    }

    //// Combine
    GpuTrackingVector<VciGpu> stepRevs;
    VciGpu bestUnsat = linkage.GetClauseCount() + 1;
    GpuTrackingVector<VciGpu> bestRevVars;
    // Make sure the overhead of preparing the combinations doesn't outnumber the effort spent in combinations
    uint32_t endComb = max(cCombsPerStep, varFront.count_ + combClauses.count_ + totListLen);
    if(varFront.count_ <= 31) {
      endComb = min(endComb, (1u<<varFront.count_)-1);
    }
    // Initial assignment
    uint32_t curComb = 1;
    {
      const VciGpu aVar = varFront.items_[0];
      stepRevs.Add<false>(aVar);
      curExec.next_.Flip(aVar);
      UpdateUnsatCs(linkage, aVar, curExec.next_, curExec.unsatClauses_);
    }
    // The first index participating in combinations - upon success, can be shifted
    VciGpu combFirst = 0;
    while(curComb <= endComb) {
      if(!trav->IsSeenAsg(curExec.next_)) {
        if(curExec.unsatClauses_.count_ < bestUnsat) {
          bestUnsat = curExec.unsatClauses_.count_;
          bestRevVars = stepRevs;
          if(bestUnsat < nStartUnsat) {
            const VciGpu oldMin = atomicMin_system(pnGlobalUnsat, bestUnsat);
            if(oldMin > bestUnsat) {
              combFirst = combFirst + __log2f(curComb-1) + 1;
              curComb = 0;
              const VciGpu remVF = varFront.count_ - combFirst;
              if(remVF <= 31) {
                endComb = min(endComb, (1u<<remVF)-1);
              }
            }
          }
        }
        if(curExec.unsatClauses_.count_ <= *pnGlobalUnsat) {
          trav->RecordAsg(curExec.next_, bestUnsat);
        }
      }
      for(uint8_t i=0; ; i++) {
        curComb ^= 1ULL << i;
        const VCIndex aVar = varFront.items_[i+combFirst];
        stepRevs.Flip(aVar);
        curExec.next_.Flip(aVar);
        UpdateUnsatCs(linkage, aVar, curExec.next_, curExec.unsatClauses_);
        if( (curComb & (1ULL << i)) != 0 ) {
          break;
        }
      }
    }
    // Check the combinations results
    if(bestUnsat > linkage.GetClauseCount()) {
      if(trav.StepBack(curExec.next_, curExec.unsatClauses_, linkage, linkage.GetClauseCount())) {
        continue;
      }
      // Increment the unsatisfied executors counter
      atomicAdd_system(pnUnsatExecs, 1);
      // The current executor considers it unsatisfiable, but let's wait for the rest of executors
      break;
    }
    if(*pnGlobalUnsat < nStartUnsat) {
      break; // some other executor found an improvement
    }
  }
}

int main(int argc, char* argv[]) {
  auto tmStart = std::chrono::steady_clock::now();
  const auto tmVeryStart = tmStart;

  if(argc < 3) {
    std::cerr << "Usage: " << argv[0] << " <input.dimacs> <output.dimacs> [<RamGBs>]" << std::endl;
    return 1;
  }

  uint64_t maxRamBytes = 0;
  if(argc >= 4) {
    maxRamBytes = std::stoull(argv[3]) * 1024 * uint64_t(1024) * 1024;
  }
  if(maxRamBytes == 0) {
    maxRamBytes = GetTotalSystemMemory() * 0.95;
  }
  
  // TODO: does it override the environment variable?
  omp_set_num_threads(nSysCpus);
  const uint64_t nOmpThreads = omp_get_max_threads();

  Formula formula;
  std::atomic<bool> provenUnsat = false;
  std::atomic<bool> maybeSat = formula.Load(argv[1]);
  if(!maybeSat) {
    provenUnsat = true;
    { // TODO: remove code duplication
      std::ofstream ofs(argv[2]);
      ofs << "s UNSATISFIABLE" << std::endl;
      // TODO: output the proof: proof.out, https://satcompetition.github.io/2024/output.html
    }
    return 0;
  }
  int64_t prevNUnsat = formula.nClauses_;

  std::cout << "Precomputing..." << std::endl;
  int nGpus = 0;
  gpuErrchk(hipGetDeviceCount(&nGpus));
  std::vector<CudaAttributes> cas(nGpus);
  std::vector<HostLinkage> linkages(nGpus);
  HostDeque<GpuPartSol> dfsPartial;
  dfsPartial.Init( maxRamBytes / 2 / (DivUp(formula.nVars_, 32)*4 + sizeof(GpuPartSol)) );
  HostRainbow hRainbow; // must be one per device
  // Pinned should be better than managed here, because managed memory transfers at page granularity,
  // while Pinned - at PCIe bus granularity, which is much smaller.
  CudaArray<GpuTraversal> trav(1, CudaArrayType::Pinned);
  for(int i=0; i<nGpus; i++) {
    cas[i].Init(i);
    // TODO: compute linkages on the CPU once, rather than building it again and again for every GPU
    linkages[i].Init(formula, cas[i]);
    dfses[i].Init()
    travs[i] = CudaArray<GpuTraversal>(1, CudaArrayType::Managed);
    travs[i].Get()->
    seenAsgs[i].Init(cas[i].freeBytes_, cas[i]);
  }
  GpuCalcHashSeries(std::max(formula.nVars_, formula.nClauses_), cas);


  return 0;
}
